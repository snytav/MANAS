#include "hip/hip_runtime.h"
template <template <class Particle> class Cell >
__global__ void GPU_eme(
		            Plasma<Cell> *gp,
		            Cell<Particle>  **cells,
		            int i_s,int l_s,int k_s,
					double *E,double *H1, double *H2,
					double *J,double c1,double c2, double tau,
					int dx1,int dy1,int dz1,int dx2,int dy2,int dz2
		)
{
	unsigned int nx = blockIdx.x;
	unsigned int ny = blockIdx.y;
	unsigned int nz = blockIdx.z;
	Cell<Particle>  *c0 = cells[0];




	gp->emeElement(*c0,i_s+nx,l_s+ny,k_s+nz,E,H1,H2,
			    	  		J,c1,c2,tau,
			    	  		dx1,dy1,dz1,dx2,dy2,dz2);
}


