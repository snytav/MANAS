#include "xeon_plasma.h"
#include <stdlib.h>
#include "mpi_shortcut.h"

int main(int argc,char*argv[])
{
      XeonPlasma<GPUCell,DIMENSIONS> *plasma;

      InitMPI(argc,argv);

#ifdef __HIPCC__
      size_t sizeP;

      printf("oarticle size %d %d \n",sizeof(Particle),sizeof(Particle)/sizeof(double));
      hipDeviceGetLimit(&sizeP,hipLimitPrintfFifoSize);

      printf("printf default limit %d \n",sizeP/1024/1024);

      sizeP *= 10;
      sizeP *= 10;
      sizeP *= 10;
      sizeP *= 10;
      hipDeviceSetLimit(hipLimitPrintfFifoSize, sizeP);

      hipDeviceGetLimit(&sizeP,hipLimitPrintfFifoSize);



      printf("printf limit set to %d \n",sizeP/1024/1024);
#endif

      int beam_plasma,start_from_file;
      double tex0,tey0,tez0,Tb,rimp,rbd,ni,lx,ly,lz,tau,B0,bx,by,bz,pl_y,pl_z;
      int lp,nx,ny,nz,np,total_steps,minor_steps,phase;

   readParameterFile(&beam_plasma,
		             &start_from_file,
		             &tex0,&tey0,&tez0,
                     &Tb,&rimp,&rbd,&ni,&lx,&ly,&lz,&lp,&nx,&ny,&nz,
	                 &tau,&B0,&np,&bx,&by,&bz,&pl_y,&pl_z,
	                 &total_steps,&minor_steps,&phase);



   int err = SetDevice(0);
   
      printf("err %d \n",err);
//   int nx0 = 100,ny0 = 4;
//   double hx = 1.1424/nx0,hy = 0.2/ny;
//
//   lx = nx*hx;
//   ly = ny*hy;
   //plasma = new GPUPlasma<GPUCell>(100,4,4,1.2566,0.05,0.05,1.0,100,1.0,0.001);
   plasma = new XeonPlasma<GPUCell,DIMENSIONS>(nx,ny,nz,lx,ly,lz,np,1.0,tau,
		    (DIMENSIONS == 3),B0,80000,
		    tex0,tey0,tez0,Tb,rimp,rbd,ni,bx,by,bz,
		    beam_plasma,start_from_file
		    );

   plasma->SetNumberOfSteps(total_steps,minor_steps,phase);
   plasma->SetPlasmaSize(pl_y,pl_z);

   plasma->Initialize();




   double t = plasma->compareCPUtoGPU();
   printf("----------------------------------------------------------- plasma check before move %.5f\n",t);
   size_t m_free,m_total;

   GetDeviceMemory(&m_free,&m_total);
   struct sysinfo info;
   int start_nt;

   if(START_STEP_NUMBER > 0)
   {
	   start_nt = START_STEP_NUMBER;
   }
   else
   {
	   start_nt = 1;
   }
//   plasma->Diagnose(START_STEP_NUMBER-1);

//   plasma->Bx0 = 0.1;

   for(int nt = START_STEP_NUMBER;nt <= total_steps;nt++)
   {
	   GetDeviceMemory(&m_free,&m_total);
	   sysinfo(&info);
#ifdef MEMORY_PRINTS
       printf("before Step  %10d CPU memory free %10u GPU memory total %10d free %10d\n",
    		   nt,info.freeram/1024/1024,m_total/1024/1024,m_free/1024/1024);
#endif

       plasma->Step(nt);
//       exit(0);
//       plasma->BeamInput(nt);

//       plasma->Diagnose(nt);


       GetDeviceMemory(&m_free,&m_total);
       sysinfo(&info);
#ifdef MEMORY_PRINTS
       printf("after  Step  %10d CPU memory free %10u GPU memory total %10d free %10d\n",
    		   nt,info.freeram/1024/1024,m_total/1024/1024,m_free/1024/1024);
#endif
   }
   exit(0);
   t = plasma->compareCPUtoGPU();
   printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ plasma check after move %.5f\n",t);

   delete plasma;

   CloseMPI();

   return 0;
}
